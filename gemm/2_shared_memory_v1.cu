#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

float testError(
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
        dim3 gridDim, dim3 blockDim, const int M, const int N, const int K);
float testPerformance(
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
        dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat);

void cpuSgemm(
        float *a, float *b, float *c, const int M, const int N, const int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}


__global__ void sgemm_V1(
        float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
        const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;  // tid/2, row of s_a
    int load_a_smem_k = (tid & 1) << 2;  // (tid % 2 == 0) ? 0 : 4, col of s_a
    int load_b_smem_k = tid >> 5;   // tid/32, row of s_b
    int load_b_smem_n = (tid & 31) << 2;  // (tid % 32) * 4, col of s_b

    int load_a_gmem_m = by * BM + load_a_smem_m;  // global row of a
    int load_b_gmem_n = bx * BN + load_b_smem_n;  // global col of b

    for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
        int load_a_gmem_k = bk * BK + load_a_smem_k;   // global col of a
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr]);
        int load_b_gmem_k = bk * BK + load_b_smem_k;   // global row of b
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr]);

        __syncthreads();

#pragma unroll
        for (int k = 0; k < BK; k++) {
#pragma unroll
            for (int m = 0; m < TM; m++) {
#pragma unroll
                for (int n = 0; n < TN; n++) {
                    int comp_a_smem_m = ty * TM + m;
                    int comp_b_smem_n = tx * TN + n;
                    r_c[m][n] += s_a[comp_a_smem_m][k] * s_b[k][comp_b_smem_n];
                }
            }
        }

        __syncthreads();
    }

#pragma unroll
    for (int i = 0; i < TM; i++) {
        int store_c_gmem_m = by * BM + ty * TM + i;
#pragma unroll
        for (int j = 0; j < TN; j += 4) {
            int store_c_gmem_n = bx * BN + tx * TN + j;
            int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
            FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][j]);
        }
    }
}

int main(void) {
    printf("\nKernal = sgemm_V1\n");
    const int outer_repeat = 10, inner_repeat = 1;
    const int BM = 128, BN = 128, TM = 8, TN = 8;
    void (*gpuSgemm) (float *, float *, float *, const int, const int, const int) = sgemm_V1;

    {
        const int M = 512, N = 512, K = 512;
        dim3 blockDim(BN / TN, BM / TM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
        float max_error = testError(gpuSgemm, gridDim, blockDim, M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};

    const int TESTNUM = 15;
    for (int i = 0; i < TESTNUM; i++) {
        const int M = M_list[i], N = N_list[i], K = K_list[i];

        dim3 blockDim(BN / TN, BM / TM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }
    return 0;
}


float testError(
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
        dim3 gridDim, dim3 blockDim, const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}


float testPerformance(
        void (*gpuSgemm) (float *, float *, float *, const int, const int, const int),
        dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat) {

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (int i = 0; i < repeat; i++)
        gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}

/*
 *Kernal = sgemm_V1
Max Error = 0.000046
M N K =    128    128   1024, Time =   0.00041882   0.00042049   0.00042630 s, AVG Performance =    74.3187 Gflops
M N K =    192    192   1024, Time =   0.00042086   0.00042179   0.00042291 s, AVG Performance =   166.7020 Gflops
M N K =    256    256   1024, Time =   0.00041885   0.00041995   0.00042086 s, AVG Performance =   297.6576 Gflops
M N K =    384    384   1024, Time =   0.00042086   0.00042168   0.00042291 s, AVG Performance =   666.9750 Gflops
M N K =    512    512   1024, Time =   0.00042086   0.00042281   0.00042598 s, AVG Performance =  1182.5743 Gflops
M N K =    768    768   1024, Time =   0.00043008   0.00044870   0.00045258 s, AVG Performance =  2507.2208 Gflops
M N K =   1024   1024   1024, Time =   0.00045053   0.00045404   0.00045568 s, AVG Performance =  4404.9137 Gflops
M N K =   1536   1536   1024, Time =   0.00074650   0.00074864   0.00075059 s, AVG Performance =  6010.8997 Gflops
M N K =   2048   2048   1024, Time =   0.00147043   0.00147395   0.00147664 s, AVG Performance =  5427.6087 Gflops
M N K =   3072   3072   1024, Time =   0.00264294   0.00282112   0.00294195 s, AVG Performance =  6380.4448 Gflops
M N K =   4096   4096   1024, Time =   0.00459162   0.00460073   0.00461926 s, AVG Performance =  6955.4184 Gflops
M N K =   6144   6144   1024, Time =   0.00981197   0.00983767   0.00985600 s, AVG Performance =  7318.8061 Gflops
M N K =   8192   8192   1024, Time =   0.01701171   0.01702492   0.01703629 s, AVG Performance =  7518.3901 Gflops
M N K =  12288  12288   1024, Time =   0.03820954   0.03828142   0.03834470 s, AVG Performance =  7523.2318 Gflops
M N K =  16384  16384   1024, Time =   0.06766899   0.06772685   0.06777139 s, AVG Performance =  7559.7791 Gflops
 */
