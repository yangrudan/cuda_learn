#include <iostream>
#include <hip/hip_runtime.h>

const int N = 100;


// Kernel definition
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < N)
        C[i][j] = A[i][j] + B[i][j];
}

int main() {
    float A[N][N], B[N][N], C[N][N];

    // Initialize matrices A and B (you may use your own initialization logic)
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i][j] = 1.0f;  // You can replace this with your initialization logic
            B[i][j] = 2.0f;  // You can replace this with your initialization logic
        }
    }

    float (*d_A)[N], (*d_B)[N], (*d_C)[N];

    // Allocate device memory
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // Kernel invocation with one block of N x N x 1 threads
    dim3 threadsPerBlock(10, 10);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;

    // Copy result from device to host
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the result (optional)
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
