#include <iostream>
#include <hip/hip_runtime.h>



int main() {
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    // do sth...

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;
}


