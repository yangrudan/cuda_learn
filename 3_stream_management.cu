#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void myKernel()
{

}


int main() {
    //创建两个流
    hipStream_t stream[2];
    for(int i = 0; i < 2; ++i){
        hipStreamCreate(&stream[i]);
    }

    //每个流 host2device, kernel run, device2host
    for(int i = 0; i < 2; ++i){
        hipMemcpyAsync(dev_input + i *size, hostPtr + i *size, size, hipMemcpyHostToDevice, stream[i]);
    }
    for(int i = 0; i < 2; ++i){
        myKernel<<<100, 512>>>(dev_out + i * size, dev_input + i *size, size)
    }
    for(int i = 0; i < 2; ++i){
        hipMemcpyAsync(dev_input + i *size, hostPtr + i *size, size, hipMemcpyDeviceToHost, stream[i]);
    }
    hipDeviceSynchronize();
}


