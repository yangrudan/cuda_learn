#include <iostream>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                          \
        printf("coda: %d, reason: %s\n", error, hipGetErrorString(error));    \
        exit(1);                                                               \
    }                                                                          \
}                                                                              \


double cpuSecond(){
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

__global__ void Check(){
    //Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    //thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    printf("threadIdx:(%d, %d, %d) | blockIdx:(%d, %d, %d) | blockDim: (%d, %d, %d) | gridDim(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);

}

int main(){
    int nElem = 6;

    dim3 dimBlock(3);
    dim3 dimGrid((nElem + dimBlock.x - 1)/dimBlock.x);

    printf("grid.x %d; grid.y %d; grid.z %d; \n", dimGrid.x, dimGrid.y, dimGrid.z);
    printf("block.x %d; block.y %d; block.z %d; \n", dimBlock.x, dimBlock.y, dimBlock.z);

    double iStart = cpuSecond();
    Check<<<dimGrid, dimBlock>>>();
    CHECK(hipDeviceReset());
    double iElaps = cpuSecond() - iStart;
}